#include "hip/hip_runtime.h"
/* 
 Author: Derek Gordon
 Date: 7/22/2013
 Project: Netflix problem/Matrix Completion
 Purpose: This project is used to test matrix completion on a dataset of movie ratings
 the dataset is split up into 2 sets.  The training set which is 80% of the movie ratings
 and the testing set which is the remaining 20%.  The algorithm will learn from the training
 set and use the testing set to test its learning when trying to predict the its own testing set.
*/

#include <iostream>
#include <math.h>
#include <string>
#include <fstream>
#include "Eigen/Core"
#include "pst.h"
#include "MovieRating.h"

using namespace Eigen;
using namespace std;

const int NUM_USERS = 943;
const int NUM_MOVIES = 1682;
const int NUM_FEATURES = 10;
const int NUM_RATINGS = 100000;
const int TEST_SIZE = 20000;
const int NUM_ITERATIONS = 2;

MovieRating StringToRating(/* in */string line);

string itos(/* in */int n);

int strtoi (/* in */const string& str);

bool IsWhiteSpace(/* in */ char c);

float RandomGuess(/* in */ const MatrixXf & p, //the user matrix
				 /* out*/ const MatrixXf & q, //the movie matrix
				 /* in */ const MatrixXf & R);

float RandomInitLearning(/* in */ const MatrixXf & p, //the user matrix
				 		 /* out*/ const MatrixXf & q, //the movie matrix
				 		 /* in */ const MatrixXf & R);

void RemoveRow(/* in */ MatrixXf & orig,
				   /* in */ const int index,
				   /* in */ const int mSize);

float RMSECalc(/* in */ const MatrixXf & rExpected,
			   /* in */ const MatrixXf & rTest);

int main(){
	MatrixXf p(NUM_USERS, NUM_FEATURES);
	MatrixXf q(NUM_MOVIES, NUM_FEATURES);
	MatrixXf R(NUM_USERS, NUM_MOVIES);
	R.setConstant(-1.f);
	MatrixXf RT(NUM_USERS, NUM_MOVIES);
	RT.setConstant(-1.f);
	
	//load training dataset
	ifstream base("assets/u1.base");
	if(base.fail()){
		cout << "Unable to open File. Program Terminating..."<<endl;
		return 1;
	}//end if
	string line = "";
	getline(base, line);
	MovieRating m = StringToRating(line);

	for(int i = 0; i < NUM_USERS; i++){
		while(m.uId == i){
			R(i,(m.mId-1)) = m.rating;
			getline(base,line);
			m = StringToRating(line);
		}//end while
	}//end for(i)

	ifstream test("assets/u1.test");
	if(base.fail()){
		cout << "Unable to open File. Program Terminating..." << endl;
		return 1;
	}

	getline(test, line);
	m = StringToRating(line);
	//load testing dataset
	for(int i = 0; i < NUM_USERS; i++){
		while(m.uId == i){
			RT(i,(m.mId-1)) = m.rating;
			getline(test,line);
			m = StringToRating(line);
		}//end while
	}//end for(i)


	float bestRMSE[NUM_ITERATIONS];
	for(int n = 0; n < NUM_ITERATIONS; n++){
		for(int i = 0; i < q.rows(); i++)
			for(int j = 0; j < q.cols(); j++)
				q(i,j) = rand() % 3;
		for(int i = 0; i < p.rows(); i++)
			for(int j = 0; j < p.cols(); j++)
				p(i,j) = rand() % 3;

		bestRMSE[n] = RandomGuess(p, q, RT);
	}
	for(int n = 0; n < NUM_ITERATIONS; n++)
		cout << bestRMSE[n] << endl;

	return 0;
}

//converts a given string to an int
int strtoi (/* in */const string& str) {
    stringstream ss(str);
    int n;
    ss >> n;
    return n;
}//end strtoi

//converts a given int to a string
string itos(/* in */int n){
    stringstream ss;
    ss << n;
    return ss.str();
}//end itos

//checks given character to see if it is whitespace
bool IsWhiteSpace(/* in */ char c){
    switch(c){
    case ' ':
    case '\t':
    case '\r':
    case '\n': return true;
    default: return false;
    }//end switch
}//end IsWhiteSpace

//Pre: string contains a properly formatted line from a dataset file
//Post: A populated MovieRating object is returned
//Purpose: to convert a string that is properly formatted to a MovieRating object
MovieRating StringToRating(/* in */string line){
    MovieRating rating;	//to process string into
    int numCat = 0; //tells which catagory to input in
    string item;
	//process line
    for(int i = 0; i <= line.size(); i++){
		//whitespace signifies end of item
        if(!IsWhiteSpace(line[i]) && i != line.size())
            item += line[i];
        else{
			//places item in the correct catagory
            if(item.size() > 0){
                switch(numCat){
                    case 0: rating.uId = strtoi(item); numCat++; break;
                    case 1: rating.mId = strtoi(item); numCat++; break;
                    case 2: rating.rating = strtoi(item); numCat++; break;
                    case 3: rating.timestamp = strtoi(item); numCat++; break;
                    default: break;
                }//end switch
                item = "";
            }//end if
        }//end else
    }//end for
    return rating;
}//end StringToRating

float RandomGuess(/* in */ const MatrixXf & p, //the user matrix
				 /* out*/ const MatrixXf & q, //the movie matrix
				 /* in */ const MatrixXf & R){ //the training matrix

	MatrixXf rTest(NUM_USERS, NUM_MOVIES);	
	MatrixXf qTemp = q;
	MatrixXf invResult(NUM_FEATURES, NUM_FEATURES);
	rTest.setConstant(-1.f);

	int mSize = NUM_MOVIES;

	for(int i = 0; i < NUM_USERS; i++){
		cout << "On Row " << i << " of " << NUM_USERS - 1 << endl;

		for(int j = NUM_MOVIES-1; j >= 0; j--)
			if(R(i,j) == -1 && mSize > 1){
				RemoveRow(qTemp, j, mSize);
				mSize--;
			}
		if(mSize >= 1){
			VectorXf pRow = p.row(i);
			VectorXf r = (q*pRow)/5;
			int index = 0;
			
			for( int k = 0; k < NUM_MOVIES; k++){
				if(R(i,k) != -1){
					rTest(i,k) = r(index);
					index++;
				}
			}
		}
		mSize = NUM_MOVIES;
		qTemp = q;
	}

	return RMSECalc(rTest, R);
}

void RemoveRow(/* in */ MatrixXf & orig,
				   /* in */ const int index,
				   /* in */ const int mSize){

		orig.row(index) = orig.row(mSize - 1);
		orig.conservativeResize(mSize - 1, NUM_FEATURES);


}

float RMSECalc(/* in */ const MatrixXf & rExpected,
			   /* in */ const MatrixXf & rTest){
	float sum = 0;
	for(int i = 0; i < NUM_USERS; i++){
		for(int j = 0; j < NUM_MOVIES; j++){
			if(rTest(i,j) != -1){
				float rE = rExpected(i,j);
				float rT = rTest(i,j);
				sum += (rE - rT) * (rE-rT);
			}
		}
	}	
	return (float)sqrt((float)sum/(float)TEST_SIZE);
}